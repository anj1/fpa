// Copyright (c) 2024. Sean Zhang.
// Splitting the different degrees to different files to speed up compilation.
// This file is auto-generated. See "generate_kernels.py"

#include "chunk_state_launch_template.h"

template<>
void run_compute_chunk_states<cutlass::bfloat16_t, 64, 2>(Chunk_state_params &params, hipStream_t stream) {
    run_chunk_states_fwd_<cutlass::bfloat16_t, 64, 2>(params, stream);
}
